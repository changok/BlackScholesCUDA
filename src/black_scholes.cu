#include "hip/hip_runtime.h"
#include <cstddef>
#include <cassert>
#include <cmath>

#include "hip/hip_runtime.h"
#include "black_scholes.h"

const int TILE_WIDTH = 128;

__device__ static inline double black_scholes_value (const double S,
             const double E, const double r, const double sigma,
             const double T, const double random_number) {
    const double current_value = S * exp ( (r - (sigma*sigma) / 2.0) * T + 
                       sigma * sqrt (T) * random_number );
    return exp (-r * T) * 
      ((current_value - E < 0.0) ? 0.0 : current_value - E);
}

__global__ void black_scholes_kernel(const double S, const double E, 
            const double r, const double sigma, const double T,
            const long M, double* blockMeans, double* cudaTrials ) {
    
    __shared__ double trials[TILE_WIDTH];
    __shared__ double sum_of_trials[TILE_WIDTH];
    
    const unsigned int tId = threadIdx.x;

    /* Do the Black-Scholes iterations */
    // M/nthreads
    const double random_number = 1.0; 
    trials[tId] = black_scholes_value (S, E, r, sigma, T, random_number);
    
    // we need origianl trial values to calculate standard deviation
    sum_of_trials[tId] = trials[tId];

    /*
     * We scale each term of the sum in order to avoid overflow. 
     * This ensures that mean is never larger than the max
     * element of trials[0 .. M-1].
     */
    for(unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        sum_of_trials[tId] += sum_of_trials[tId + stride];
    }

    /* Pack the OUT values into the args struct */
    if(tId == 0) {
        blockMeans[blockIdx.x] = sum_of_trials[0]/(double)M;
    }
}

cit black_scholes(const double S, const double E, const double r,
                   const double sigma, const double T, const long M) {
    cit interval;
    long num_of_blocks = M/TILE_WIDTH;
    double* means = new double[num_of_blocks];
    double stddev = 0.0;
    double conf_width = 0.0;

    assert (M > 0);
    double* trials = new double[M]; //Array containing the results of each of the M trials.
    long size = M * sizeof(double);
    assert (trials != NULL);

    double* blockMeans;
    hipMalloc((void**) &blockMeans, size);
    
    double* cudaTrials;
    hipMalloc((void**) &cudaTrials, size);

    dim3 dimGrid(num_of_blocks);
    dim3 dimBlock(TILE_WIDTH);

    black_scholes_kernel<<<dimGrid, dimBlock>>>(S, E, r, sigma, T, M, blockMeans, cudaTrials);
    //black_scholes_kernel (S, E, r, sigma, T, M, cudaMean, cudaTrials);
    
    hipMemcpy(&means, &blockMeans, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(blockMeans);
    
    hipMemcpy(&trials, &cudaTrials, size, hipMemcpyDeviceToHost);
    hipFree(cudaTrials);
    
    double mean = 0.0;
    // combine results from each threads
    for (long i = 0; i < num_of_blocks; i++) {
        mean += means[i];
    }
    
    stddev = black_scholes_stddev (mean, M, trials);

    conf_width = 1.96 * stddev / sqrt ((double) M);
    interval.min = mean - conf_width;
    interval.max = mean + conf_width;

    delete [] trials;
    delete [] means;
    
    return interval;
}

/**
 * Compute the standard deviation of trials[0 .. M-1].
 */
static double black_scholes_stddev (const double mean, const long M, const double* trials) {
    double variance = 0.0;
    long k;
    
    for (k = 0; k < M; k++) {
        const double diff = trials[k] - mean;
        /*
        * Just like when computing the mean, we scale each term of this
        * sum in order to avoid overflow.
        */
        variance += diff * diff / (double) M;
    }
    
    return sqrt (variance);
}