#include "hip/hip_runtime.h"
#include <iostream>
#include <cstddef>
#include <cassert>
#include <cmath>

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "common.h"
#include "black_scholes.cuh"
#include "timer.h"

using namespace std;

__global__ void setup_rnd_kernel ( hiprandState * state, time_t seed )
{
    long id = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__device__ double black_scholes_value (const double S,
             const double E, const double r, const double sigma,
             const double T, const double random_number) {
    const double current_value = S * exp ( (r - (sigma*sigma) / 2.0) * T + 
                       sigma * sqrt (T) * random_number );
    return exp (-r * T) * 
      ((current_value - E < 0.0) ? 0.0 : current_value - E);
}

// standard normal distributed random number [0~1]
__device__ gaussrand_result_t gaussrand (hiprandState* localState) {
  gaussrand_result_t result;

  double v1, v2, s;
  do {
    v1 = 2.0 * hiprand_uniform(localState) - 1.0;
    v2 = 2.0 * hiprand_uniform(localState) - 1.0;
    s = v1 * v1 + v2 * v2;
  } while (s >= 1 || s== 0);

  double w = sqrt ( (-2.0 * log (s)) / s);

  	result.grand1 = v1 * w;
  	result.grand2 = v2 * w;

  return result;
}

__global__ void black_scholes_kernel(const double S, const double E,
            const double r, const double sigma, const double T,
            const long M, double* blockMeans, double* cudaTrials,
            hiprandState* randStates, const int mode, const double* fixedRands,
            double* debug, int debug_mode) {
    
    __shared__ double means[WINDOW_WIDTH];
   
    const long NUM_OF_TOT_THREAD = gridDim.x * blockDim.x;
    const long LOOP_SIZE = M / NUM_OF_TOT_THREAD;
    const unsigned int GID = (blockIdx.x * blockDim.x) * LOOP_SIZE + threadIdx.x * LOOP_SIZE;
    const unsigned int TID = threadIdx.x;

	
    hiprandState localState = randStates[(blockIdx.x * blockDim.x) + threadIdx.x];
    gaussrand_result_t gresult;

    means[TID] = 0.0;

    // Do the Black-Scholes iterations
    for(long trial = 0; trial < LOOP_SIZE; trial++) {
        double value = 0.0;
        if (trial%2 == 0) {
            if (mode == 1) {
                gresult.grand1 = 1.0;
                gresult.grand2 = 1.0;
            }
            // use pre-generated random number
            else if (mode == 2) {
                gresult.grand1 = fixedRands[GID + trial];
                gresult.grand2 = fixedRands[GID + trial+1];
            }
            // use gaussian random number (standard normal distributed)
            else {
                gresult = gaussrand (&localState);
            }

            if(debug_mode == 1) {
                debug[GID + trial] = gresult.grand1;
            }
            value = black_scholes_value (S, E, r, sigma, T, gresult.grand1);
        } else {
            if(debug_mode == 1) {
                debug[GID + trial] = gresult.grand2;
            }
            value = black_scholes_value (S, E, r, sigma, T, gresult.grand2);
        }

        // we need to keep origianl trial values for calculatng standard deviation
        // for current calculation, we use trials
        // Also, to prevent overflow caused by adding, divide the value by M in advance
        means[TID] += value/M;
        cudaTrials[GID + trial] = value;
    }

    for(unsigned int stride = blockDim.x>>1; stride > 0; stride >>= 1) {
        __syncthreads();
		if (TID < stride)
		    means[TID] += means[TID + stride];
    }

    if(TID == 0) {
        blockMeans[blockIdx.x] = means[0];
    }
}

__device__ void trunc(double* target) {
    if (*target < 0.0000000005 && *target > 0)
        *target = 0.0;
    if (*target > -0.0000000005 && *target < 0)
        *target = 0.0;

    //*target = *target > 0.0 ? floor(*target + 0.00000000005) : ceil(*target - 0.0000000005);
}

__global__ void black_scholes_variance_kernel(const double mean,
            const long M, double* cudaTrials, double* cudaVariances) {
    
    __shared__ double variances[WINDOW_WIDTH];
    
    unsigned int gId = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x; 
    unsigned int tId = threadIdx.x;
    
    variances[tId] = cudaTrials[gId];
    variances[tId] = variances[tId] - mean;
    trunc(&variances[tId]);
    variances[tId] = (variances[tId] *  variances[tId])/ (double)(M-1);

    for(unsigned int stride = blockDim.x>>1; stride > 0; stride >>= 1) {
        __syncthreads();
		if (stride > tId)
        	variances[tId] += variances[tId + stride];
    }

    if(tId == 0) {
        cudaVariances[blockIdx.x] = variances[0];
    }
}

cit black_scholes(const double S, const double E, const double r,
                   const double sigma, const double T, const long M,
                   const int mode, double* cudafixedRands, int debug_mode) {
    cit interval;
    const long num_of_blocks = min((M/WINDOW_WIDTH), BLOCK_SIZE);
    long num_of_tot_threads = num_of_blocks * WINDOW_WIDTH;

    double* means = new double[num_of_blocks];
    double stddev = 0.0;
    double conf_width = 0.0;
	double t1, t2;

// part1_begin
	t1 = get_seconds();

    assert (M > 0);
    long size = M * sizeof(double);

    dim3 dimGrid(num_of_blocks);
    dim3 dimBlock(WINDOW_WIDTH);

	t2 = get_seconds();
	interval.t1 = t2-t1;	// init time
// part1_end

    // below pretend working with MOCKED thread number
// part2_begin
	t1 = 0; t1 = get_seconds();
    hiprandState* randStates;
    hipMalloc((void **) &randStates, num_of_tot_threads * sizeof(hiprandState));
	t2 = 0; t2 = get_seconds();
    interval.t2 = t2- t1;	// setup_rnd_kernel time
// part2_end

// part5_begin
	t1 = 0; t1 = get_seconds();
    setup_rnd_kernel<<<dimGrid, dimBlock>>>(randStates, time(NULL));
	t2 = 0; t2 = get_seconds();
	interval.t5 = t2-t1;
// part5_end

// part3_begin
	t1 = 0; t1 = get_seconds();

    double* blockMeans;
    hipMalloc((void**) &blockMeans, num_of_blocks * sizeof(double));
    
    double* cudaTrials;
    hipMalloc((void**) &cudaTrials, size);

    double* hostDebug = new double[M];
    double* cudaDebug;
    if (debug_mode == 1) {
        hipMalloc((void**) &cudaDebug, size);
    }

    black_scholes_kernel<<<dimGrid, dimBlock>>>(S, E, r, sigma, T, M, blockMeans, cudaTrials, randStates, mode, cudafixedRands, cudaDebug, debug_mode);

    hipMemcpy(means, blockMeans, num_of_blocks * sizeof(double), hipMemcpyDeviceToHost);

    if (debug_mode == 1) {
        hipMemcpy(hostDebug, cudaDebug, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < M; i++) {
            printf("r%d: %lf, ", i, hostDebug[i]);
        }
        puts("\n");

        for (int i = 0; i < num_of_blocks; i++) {
            printf("m%d: %lf, ", i, means[i]);
        }
        puts("");

        double* t = new double[M];
        hipMemcpy(t, cudaTrials, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < M; i++) {
            printf("t%d: %lf, ", i, t[i]);
        }
        puts("");

        delete [] t;
    }

	t2 =0; t2 = get_seconds();
	interval.t3 = t2-t1;	// black_scholes_kernel time
// part3_end

// part4_begin
    t1 = 0;
	t1 = get_seconds();
    double mean = 0.0;

    // combine results from each blocks
    for (long i = 0; i < num_of_blocks; i++) {
        mean += means[i];
    }
   
	printf("mean: %.20lf\n", mean);

    stddev = black_scholes_stddev (mean, M, cudaTrials);
    cout << "StdDev: " << stddev << endl;


	t2 = 0;
	t2 = get_seconds();
	interval.t4 = t2-t1;
// part4_end

    /* confidence interval */
    conf_width = 1.96 * stddev / sqrt ((double) M);
    interval.min = mean - conf_width;
    interval.max = mean + conf_width;

    /* clean up */
	hipFree(cudaDebug);
    hipFree(cudaTrials);
    hipFree(blockMeans);
    hipFree(randStates);

    delete [] hostDebug;
    delete [] means;
    
    return interval;
}

double black_scholes_stddev (const double mean, const long M, double* cudaTrials) {
    double* variances = new double[M/WINDOW_WIDTH];
    double* cudaVariances;
    hipMalloc((void**) &cudaVariances, M/WINDOW_WIDTH * sizeof(double));
    
//    dim3 dimGrid(M/WINDOW_WIDTH);
    dim3 dimGrid(M/WINDOW_WIDTH);
    dim3 dimBlock(WINDOW_WIDTH);
    black_scholes_variance_kernel<<<dimGrid, dimBlock>>>(mean, M, cudaTrials, cudaVariances);

    hipMemcpy(variances, cudaVariances, M/WINDOW_WIDTH * sizeof(double), hipMemcpyDeviceToHost);
    
    double variance = 0.0;
    for(long idx=0; idx<M/WINDOW_WIDTH; idx++) {
        cout << "std: " << variances[idx] << ", ";
        variance += variances[idx];
    }
    cout << endl;

    hipFree(cudaVariances);
    
    return sqrt(variance);
}
