#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <cstddef>
#include <cassert>
#include <cmath>

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "black_scholes.cuh"

using namespace std;

const int WINDOW_WIDTH = 128;

__global__ void setup_rnd_kernel ( hiprandState * state, unsigned long seed )
{
    long id = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__device__ double black_scholes_value (const double S,
             const double E, const double r, const double sigma,
             const double T, const double random_number) {
    const double current_value = S * exp ( (r - (sigma*sigma) / 2.0) * T + 
                       sigma * sqrt (T) * random_number );
    return exp (-r * T) * 
      ((current_value - E < 0.0) ? 0.0 : current_value - E);
}

__global__ void black_scholes_kernel(const double S, const double E, 
            const double r, const double sigma, const double T,
            const long M, double* blockMeans, double* cudaTrials,
            hiprandState* rnds, double* prns) {
    
    __shared__ double sum_of_trials[WINDOW_WIDTH];
    
    unsigned int gId = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x; 
    unsigned int tId = threadIdx.x;

    // Do the Black-Scholes iterations
    //const double random_number = 1.0; 
    hiprandState localState = rnds[gId];
    double RANDOM = hiprand_uniform( &localState );

//test
prns[gId] = RANDOM;

    rnds[gId] = localState;

    double value = black_scholes_value (S, E, r, sigma, T, RANDOM);
    cudaTrials[gId] = value;
    
    // we need to keep origianl trial values for calculatng standard deviation
    sum_of_trials[tId] = value;

    for(unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        sum_of_trials[tId] += sum_of_trials[tId + stride];
    }

    // Pack the OUT values into the args struct 
    if(tId == 0) {
        blockMeans[blockIdx.x] = sum_of_trials[0]/(double)M;
    }
}

__global__ void black_scholes_variance_kernel(const double mean,
            const long M, double* cudaTrials, double* cudaVariances) {
    
    __shared__ double variances[WINDOW_WIDTH];
    
    unsigned int gId = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x; 
    unsigned int tId = threadIdx.x;
    
    variances[tId] = cudaTrials[gId];
    variances[tId] = variances[tId] - mean;
    variances[tId] = variances[tId] *  variances[tId] / (double)M;

    for(unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        variances[tId] += variances[tId + stride];
    }

    if(tId == 0) {
        cudaVariances[blockIdx.x] = variances[0];
    }
}

cit black_scholes(const double S, const double E, const double r,
                   const double sigma, const double T, const long M) {
    cit interval;
    int num_of_blocks = M/WINDOW_WIDTH;
    double* means = new double[num_of_blocks];
    double stddev = 0.0;
    double conf_width = 0.0;

    assert (M > 0);
    double* trials = new double[M]; //Array containing the results of each of the M trials.
    assert (trials != NULL);
    long size = M * sizeof(double);

    dim3 dimGrid(num_of_blocks);
    dim3 dimBlock(WINDOW_WIDTH);

    hiprandState* devStates;
    hipMalloc((void **) &devStates, M * sizeof(hiprandState));
    setup_rnd_kernel<<<dimGrid, dimBlock>>>(devStates, time(NULL));

    double* blockMeans;
    hipMalloc((void**) &blockMeans, num_of_blocks * sizeof(double));
    
    double* cudaTrials;
    hipMalloc((void**) &cudaTrials, size);

//test
double* prns;
hipMalloc((void**) &prns, size);
double* prns_table = new double[size];

    black_scholes_kernel<<<dimGrid, dimBlock>>>(S, E, r, sigma, T, M, blockMeans, cudaTrials, devStates, prns);
    
    hipMemcpy(means, blockMeans, num_of_blocks * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(blockMeans);

//test
hipMemcpy(prns_table, prns, size, hipMemcpyDeviceToHost);
for (long i = 0; i < M; i++) {
	printf("%lf, ", prns_table[i]);
	//cout << prns_table[i] << ", ";
}
cout << endl;
    
    double mean = 0.0;
    // combine results from each threads
    for (long i = 0; i < num_of_blocks; i++) {
        mean += means[i];
    }
    
    stddev = black_scholes_stddev (mean, M, cudaTrials);
    hipMemcpy(trials, cudaTrials, size, hipMemcpyDeviceToHost);
    hipFree(cudaTrials);

    conf_width = 1.96 * stddev / sqrt ((double) M);
    interval.min = mean - conf_width;
    interval.max = mean + conf_width;

    hipFree(devStates);
    delete [] trials;
    delete [] means;
    
    return interval;
}

double black_scholes_stddev (const double mean, const long M, double* cudaTrials) {
    const long numOfBlocks = M/WINDOW_WIDTH;
    double* variances = new double[numOfBlocks]; 
    double* cudaVariances;
    hipMalloc((void**) &cudaVariances, numOfBlocks * sizeof(double));
    
    dim3 dimGrid(numOfBlocks);
    dim3 dimBlock(WINDOW_WIDTH);
    black_scholes_variance_kernel<<<dimGrid, dimBlock>>>(mean, M, cudaTrials, cudaVariances);
    hipMemcpy(variances, cudaVariances, numOfBlocks * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(cudaVariances);
    
    double variance = 0.0;
    for(long idx=0; idx<numOfBlocks; idx++) {
        variance += variances[idx];
    }
    
    return sqrt(variance);
}

/**
 * Compute the standard deviation of trials[0 .. M-1].
 */
/*
double black_scholes_stddev (const double mean, const long M, double* trials) {
    double variance = 0.0;
    long k;
    
    for (k = 0; k < M; k++) {
        const double diff = trials[k] - mean;
        
        // Just like when computing the mean, we scale each term of this
        // sum in order to avoid overflow.
        //
        variance += diff * diff / (double) M;
    }
    
    return sqrt (variance);
}
*/
