#include "hip/hip_runtime.h"
#include <iostream>
#include <cstddef>
#include <cassert>
#include <cmath>

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "black_scholes.cuh"
#include "timer.h"

using namespace std;

const int WINDOW_WIDTH = 128;

__global__ void setup_rnd_kernel ( hiprandState * state, time_t seed )
{
    long id = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__device__ double black_scholes_value (const double S,
             const double E, const double r, const double sigma,
             const double T, const double random_number) {
    const double current_value = S * exp ( (r - (sigma*sigma) / 2.0) * T + 
                       sigma * sqrt (T) * random_number );
    return exp (-r * T) * 
      ((current_value - E < 0.0) ? 0.0 : current_value - E);
}

// standard normal distributed random number [0~1]
__device__ gaussrand_result_t gaussrand (hiprandState* localState) {
  gaussrand_result_t result;

  double v1, v2, s;
  do {
    v1 = 2.0 * hiprand_uniform(localState) - 1.0;
    v2 = 2.0 * hiprand_uniform(localState) - 1.0;
    s = v1 * v1 + v2 * v2;
  } while (s >= 1 || s== 0);

  double w = sqrt ( (-2.0 * log (s)) / s);

  	result.grand1 = v1 * w;
  	result.grand2 = v2 * w;

  return result;
}

__global__ void black_scholes_kernel(const double S, const double E, 
            const double r, const double sigma, const double T,
            const long M, double* blockMeans, double* cudaTrials,
            hiprandState* randStates, const int mode, const double* fixedRands,
			double* debug) {
    
    __shared__ double sum_of_trials[WINDOW_WIDTH];
   
    /* Since we decreased ACTUAL size of block to the half
       , gId which is thread position in KERNEL will be jumped
       by WINDOW_WIDTH/2 at each end of tId in a block */
    unsigned int gId = (blockIdx.x * WINDOW_WIDTH) + (threadIdx.x); 
    unsigned int tId = threadIdx.x;
	
	int pad = WINDOW_WIDTH/2;

    // Do the Black-Scholes iterations
    hiprandState localState = randStates[gId];
	gaussrand_result_t gresult;

	// use 1 as random number
	if (mode == 1) {
		gresult.grand1 = 1.0;
		gresult.grand2 = 1.0;
	}
	// use pre-generated random number
	else if (mode == 2) {
	    gresult.grand1 = fixedRands[gId];
		gresult.grand2 = fixedRands[gId+pad];
	}
	// use gaussian random number (standard normal distributed)
	else {
    	gresult = gaussrand (&localState);
	}

debug[gId] = gresult.grand1;
debug[gId+pad] = gresult.grand2;

	randStates[gId] = localState;

    gresult.grand1 = black_scholes_value (S, E, r, sigma, T, gresult.grand1);
    cudaTrials[gId] = gresult.grand1;
    gresult.grand2 = black_scholes_value (S, E, r, sigma, T, gresult.grand2);
    cudaTrials[gId+pad] = gresult.grand2;
    // we need to keep origianl trial values for calculatng standard deviation
    sum_of_trials[tId] = gresult.grand1;
    sum_of_trials[tId+pad] = gresult.grand2;

    // don't have to do "blockDim.x >> 1" because of faked size of block
    for(unsigned int stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
		if (tId < stride)  
		  sum_of_trials[tId] += sum_of_trials[tId + stride];
    }

    // Pack the OUT values into the args struct 
    if(tId == 0) {
        blockMeans[blockIdx.x] = sum_of_trials[0]/(double)M;
    }
}

__global__ void black_scholes_variance_kernel(const double mean,
            const long M, double* cudaTrials, double* cudaVariances) {
    
    __shared__ double variances[WINDOW_WIDTH];
    
    unsigned int gId = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x; 
    unsigned int tId = threadIdx.x;
    
    variances[tId] = cudaTrials[gId];
    variances[tId] = variances[tId] - mean;
    variances[tId] = (variances[tId] *  variances[tId] )/ (double)M;

    for(unsigned int stride = blockDim.x>>1; stride > 0; stride >>= 1) {
        __syncthreads();
		if (stride > tId)
        	variances[tId] += variances[tId + stride];
    }

    if(tId == 0) {
        cudaVariances[blockIdx.x] = variances[0];
    }
}

cit black_scholes(const double S, const double E, const double r,
                   const double sigma, const double T, const long M,
                   const int mode, double* cudafixedRands) {

    cit interval;
    int num_of_blocks = M/WINDOW_WIDTH;
    double* means = new double[num_of_blocks];
    double stddev = 0.0;
    double conf_width = 0.0;
	double t1, t2;

// part1_begin
	t1 = get_seconds();

    assert (M > 0);
    double* trials = new double[M]; //Array containing the results of each of the M trials.
    long size = M * sizeof(double);
    assert (trials != NULL);

    dim3 dimGrid(num_of_blocks);
    /* The number of threads in each block was supposed to be WINDOW_WIDTH(128). 
       However, the feature of gaussian randum number generator forced us to make 
       work per each thread to be twice of black scholes value operation instead of one. 
       So, we decide to decrease the ACTUAL number of thread(WINDOW_WIDTH) by the half of
       , and pretend to operate with WINDOW_WIDTH number of threads in a block. */
    dim3 dimBlock(WINDOW_WIDTH/2);

	t2 = get_seconds();
	interval.t1 = t2-t1;	// init time
// part1_end

    // below pretend working with MOCKED thread number

// part2_begin
	t1 = 0; t1 = get_seconds();
    hiprandState* randStates;
    hipMalloc((void **) &randStates, M * sizeof(hiprandState));
	t2 = 0; t2 = get_seconds();
    interval.t2 = t2- t1;	// setup_rnd_kernel time
// part2_end

// part5_begin
	t1 = 0; t1 = get_seconds();
    setup_rnd_kernel<<<dimGrid, dimBlock>>>(randStates, time(NULL));
	t2 = 0; t2 = get_seconds();
	interval.t5 = t2-t1;
// part5_end

// part3_begin
	t1 = 0; t1 = get_seconds();

    double* blockMeans;
    hipMalloc((void**) &blockMeans, num_of_blocks * sizeof(double));
    
    double* cudaTrials;
    hipMalloc((void**) &cudaTrials, size);


double* hostDebug = new double[M];
double* cudaDebug;
hipMalloc((void**) &cudaDebug, size);

    black_scholes_kernel<<<dimGrid, dimBlock>>>(S, E, r, sigma, T, M, blockMeans, cudaTrials, randStates, mode, cudafixedRands, cudaDebug);
	
    hipMemcpy(means, blockMeans, num_of_blocks * sizeof(double), hipMemcpyDeviceToHost);

hipMemcpy(hostDebug, cudaDebug, size, hipMemcpyDeviceToHost);
for (int i = 0; i < M; i++) {
  printf("%lf, ", hostDebug[i]);
}
puts("");

	t2 =0; t2 = get_seconds();
	interval.t3 = t2-t1;	// black_scholes_kernel time
// part3_end

// part4_begin
    t1 = 0;
	t1 = get_seconds();
    double mean = 0.0;

    // combine results from each threads
    for (long i = 0; i < num_of_blocks; i++) {
        mean += means[i];
    }
   
	printf("mean: %.20lf\n", mean);

    stddev = black_scholes_stddev (mean, M, cudaTrials);

    hipMemcpy(trials, cudaTrials, size, hipMemcpyDeviceToHost);

	t2 = 0;
	t2 = get_seconds();
	interval.t4 = t2-t1;
// part4_end

    /* confidence interval */
    conf_width = 1.96 * stddev / sqrt ((double) M);
    interval.min = mean - conf_width;
    interval.max = mean + conf_width;

    /* clean up */
	hipFree(cudaDebug);
	delete [] hostDebug;

    hipFree(cudaTrials);
    hipFree(blockMeans);
    hipFree(randStates);
    delete [] trials;
    delete [] means;
    
    return interval;
}

double black_scholes_stddev (const double mean, const long M, double* cudaTrials) {
    const long num_of_blocks = max((long)1,M/WINDOW_WIDTH);
    double* variances = new double[num_of_blocks]; 
    double* cudaVariances;
    hipMalloc((void**) &cudaVariances, num_of_blocks * sizeof(double));
    
    dim3 dimGrid(num_of_blocks);
    dim3 dimBlock(WINDOW_WIDTH);
    black_scholes_variance_kernel<<<dimGrid, dimBlock>>>(mean, M, cudaTrials, cudaVariances);
    hipMemcpy(variances, cudaVariances, num_of_blocks * sizeof(double), hipMemcpyDeviceToHost);
    
    double variance = 0.0;
    for(long idx=0; idx<num_of_blocks; idx++) {
        variance += variances[idx];
    }

    /* clean up */
    hipFree(cudaVariances);
    
    return sqrt(variance);
}
