#include "hip/hip_runtime.h"
#include <iostream>
#include <cstddef>
#include <cassert>
#include <cmath>

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "cutil.h"
#include "cutil_inline_runtime.h"

#include "common.h"
#include "black_scholes.cuh"
#include "timer.h"
#include "bsconfig.h"

using namespace std;

__global__ void setup_rnd_kernel ( hiprandState * state, time_t seed )
{
    long id = (blockIdx.x * WINDOW_WIDTH) + threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__device__ double black_scholes_value (BSConfig cf, const double random_number) {
    const double current_value = cf.S * exp ( (cf.r - (cf.sigma*cf.sigma) / 2.0) * cf.T +
                                 cf.sigma * sqrt (cf.T) * random_number );
    return exp (-cf.r * cf.T) *
      ((current_value - cf.E < 0.0) ? 0.0 : current_value - cf.E);
}

// standard normal distributed random number [0~1]
__device__ gaussrand_result_t gaussrand (hiprandState* localState) {
  gaussrand_result_t result;

  double v1, v2, s;
  do {
    v1 = 2.0 * hiprand_uniform(localState) - 1.0;
    v2 = 2.0 * hiprand_uniform(localState) - 1.0;
    s = v1 * v1 + v2 * v2;
  } while (s >= 1 || s== 0);

  double w = sqrt ( (-2.0 * log (s)) / s);

  	result.grand1 = v1 * w;
  	result.grand2 = v2 * w;

  return result;
}

#ifdef __GOGO_DEBUG__
__global__ void black_scholes_kernel(double* blockMeans, double* cudaTrials,
            hiprandState* randStates, const double* fixedRands, double* debug, BSConfig config) {
#else
    __global__ void black_scholes_kernel(double* blockMeans, double* cudaTrials,
                hiprandState* randStates, BSConfig config) {
#endif
    
    __shared__ double means[WINDOW_WIDTH];
   
    const long LOOP_SIZE = (long)1 < (config.M / (BLOCK_SIZE * WINDOW_WIDTH)) ? config.M / (BLOCK_SIZE * WINDOW_WIDTH) : 1;
    const unsigned int GID = (blockIdx.x * blockDim.x) * LOOP_SIZE + threadIdx.x * LOOP_SIZE;
    const unsigned int TID = threadIdx.x;
	
    hiprandState localState = randStates[(blockIdx.x * blockDim.x) + threadIdx.x];
    gaussrand_result_t gresult;

    means[TID] = 0.0;

    // Do the Black-Scholes iterations
    for(long trial = 0; trial < LOOP_SIZE; trial++) {
        double value = 0.0;
        if (trial%2 == 0) {
            if (config.RND_MODE == 1) {
                gresult.grand1 = 1.0;
                gresult.grand2 = 1.0;
            }
#ifdef __GOGO_DEBUG__
            // use pre-generated random number
            else if (config.RND_MODE == 2) {
                gresult.grand1 = fixedRands[GID + trial];
                gresult.grand2 = fixedRands[GID + trial+1];

                if(config.DEBUG_LEVEL == 2) {
                    debug[GID + trial] = gresult.grand1;
                    debug[GID + trial + 1] = gresult.grand2;
                }
            }
#endif
            // use gaussian random number (standard normal distributed)
            else {
                gresult = gaussrand (&localState);
            }

            value = black_scholes_value (config, gresult.grand1);
        } else {
            value = black_scholes_value (config, gresult.grand2);
        }

        // we need to keep origianl trial values for calculatng standard deviation
        // for current calculation, we use trials
        // Also, to prevent overflow caused by adding, divide the value by M in advance
        means[TID] += value/config.M;
        cudaTrials[GID + trial] = value;
    }

    for(unsigned int stride = blockDim.x>>1; stride > 0; stride >>= 1) {
        __syncthreads();
		if (TID < stride)
		    means[TID] += means[TID + stride];
    }

    if(TID == 0) {
        blockMeans[blockIdx.x] = means[0];
    }
}

__device__ void trunc(double* target) {
    if (*target < 0.0000000005 && *target > 0)
        *target = 0.0;
    else if (*target > -0.0000000005 && *target < 0)
        *target = 0.0;
}

#ifdef __GOGO_DEBUG__
__global__ void black_scholes_variance_kernel(const long M, const double mean,
            double* cudaTrials, double* cudaVariances, double* debug) {
#else
__global__ void black_scholes_variance_kernel(const long M, const double mean,
            double* cudaTrials, double* cudaVariances) {
#endif

        __shared__ double variances[WINDOW_WIDTH];
    
    const long LOOP_SIZE = (long)1 < (M / (BLOCK_SIZE * WINDOW_WIDTH)) ? M / (BLOCK_SIZE * WINDOW_WIDTH) : 1;
    const unsigned int GID = (blockIdx.x * blockDim.x) * LOOP_SIZE + threadIdx.x * LOOP_SIZE;
    const unsigned int TID = threadIdx.x;

    // Do the Black-Scholes iterations
    variances[TID] = 0;
    for(long trial = 0; trial < LOOP_SIZE; trial++) {
        double v = cudaTrials[GID + trial];
        v = v - mean;

        // Meaningless value such as 1.1E-15 could lead invalid result
        // when number of trial is so high. Thus, truncate all after the 10th
        // decimal place. Even though we truncate them, the result still in
        // acceptable valid range
        trunc(&v);

        variances[TID] += (v *  v) / (double)(M-1);
#ifdef __GOGO_DEBUG__
        debug[GID + trial] = v;
#endif
    }
    
    for(unsigned int stride = WINDOW_WIDTH>>1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (stride > TID)
            variances[TID] += variances[TID + stride];
    }

    if(TID == 0) {
        cudaVariances[blockIdx.x] = variances[0];
    }
}

#ifdef __GOGO_DEBUG__
Result black_scholes(double* cudafixedRands, BSConfig config) {
#else
Result black_scholes(BSConfig config) {
#endif
    Result result;
    double* means = new double[config.totalNumOfBlocks()];
    double conf_width = 0.0;
	double t1, t2;

    assert (config.M > 0);
    long size = config.M * sizeof(double);

    dim3 dimGrid(config.totalNumOfBlocks());
    dim3 dimBlock(WINDOW_WIDTH);

    // part5_start
	t1 = get_seconds();
    hiprandState* randStates;
    cutilSafeCall(hipMalloc((void **) &randStates, config.totalNumOfThread() * sizeof(hiprandState)));

    setup_rnd_kernel<<<dimGrid, dimBlock>>>(randStates, time(NULL));
	t2 = get_seconds();
	result.init_seeds_setup_time = t2 - t1;
	// part5_end

	// part3_begin
	t1 = get_seconds();

    double* blockMeans;
    cutilSafeCall(hipMalloc((void**) &blockMeans, config.totalNumOfBlocks() * sizeof(double)));
    
    double* cudaTrials;
    cutilSafeCall(hipMalloc((void**) &cudaTrials, size));
#ifdef __GOGO_DEBUG__
    double* hostDebug = new double[config.M];
    double* cudaDebug;
    cutilSafeCall(hipMalloc((void**) &cudaDebug, size));

    black_scholes_kernel<<<dimGrid, dimBlock>>>(blockMeans, cudaTrials, randStates, cudafixedRands, cudaDebug, config);
#else
    black_scholes_kernel<<<dimGrid, dimBlock>>>(blockMeans, cudaTrials, randStates, config);
#endif

    cutilSafeCall(hipMemcpy(means, blockMeans, config.totalNumOfBlocks() * sizeof(double), hipMemcpyDeviceToHost));

#ifdef __GOGO_DEBUG__
    if (config.DEBUG_LEVEL == 2) {
        hipMemcpy(hostDebug, cudaDebug, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < config.M; i++) {
            if(i < 10 || i > (config.M - 10))
                printf("RND[%d]: %lf\n", i, hostDebug[i]);
        }
        puts("\n");

        for (int i = 0; i < config.totalNumOfBlocks(); i++) {
            if(i < 10 || i > (config.M - 10))
                printf("MEAN[%d]: %lf\n", i, means[i]);
        }
        puts("");

        double* t = new double[config.M];
        cutilSafeCall(hipMemcpy(t, cudaTrials, size, hipMemcpyDeviceToHost));
        for (int i = 0; i < config.M; i++) {
            if(i < 10 || i > (config.M - 10))
                printf("TRIAL[%d]: %lf\n", i, t[i]);
        }
        puts("");

        delete [] t;
    }
#endif

	t2 = get_seconds();
	result.black_sholes_kernel_time = t2 - t1;
	// part3_end

	// part4_begin
	t1 = get_seconds();
    result.mean = 0.0;

    // combine results from each blocks
    for (long i = 0; i < config.totalNumOfBlocks(); i++) {
        result.mean += means[i];
    }

	result.stddev = black_scholes_stddev(result.mean, config, cudaTrials);

	t2 = get_seconds();
	result.calc_stddev_time = t2 - t1;
	// part4_end

    // confidence interval
    conf_width = 1.96 * result.stddev / sqrt ((double) config.M);
    result.min = result.mean - conf_width;
    result.max = result.mean + conf_width;

    /* clean up */
#ifdef __GOGO_DEBUG__
    hipFree(cudaDebug);
#endif
    hipFree(cudaTrials);
    hipFree(blockMeans);
    hipFree(randStates);

#ifdef __GOGO_DEBUG__
    if(hostDebug != NULL) delete [] hostDebug;
#endif
    if(means != NULL) delete [] means;

    return result;
}

double black_scholes_stddev (const double mean, BSConfig config, double* cudaTrials) {
    double* variances = new double[config.totalNumOfBlocks()];
    double* cudaVariances;
    cutilSafeCall(hipMalloc((void**) &cudaVariances, (config.totalNumOfBlocks()) * sizeof(double)));

    dim3 dimGrid(config.totalNumOfBlocks());
    dim3 dimBlock(WINDOW_WIDTH);

    double variance = 0.0;
#ifdef __GOGO_DEBUG__
    double* debug = new double[config.M];

    double* cudaDebug;
    cutilSafeCall(hipMalloc((void**) &cudaDebug, config.M * sizeof(double)));
    black_scholes_variance_kernel<<<dimGrid, dimBlock>>>(config.M, mean, cudaTrials, cudaVariances, cudaDebug);

    cutilSafeCall(hipMemcpy(variances, cudaVariances, config.totalNumOfBlocks() * sizeof(double), hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(debug, cudaDebug, config.M * sizeof(double), hipMemcpyDeviceToHost));

    for(long idx=0; idx<config.M; idx++) {
        if(config.DEBUG_LEVEL == 2) {
            if(idx < 10 || idx > (config.M - 10))
                cout << "THR_VAR[" << idx << "]: " << debug[idx] << endl;
        }
    }
    cout << endl;

    hipFree(cudaDebug);
    delete [] debug;
#else
    black_scholes_variance_kernel<<<dimGrid, dimBlock>>>(config.M, mean, cudaTrials, cudaVariances);
#endif

    cutilSafeCall(hipMemcpy(variances, cudaVariances, config.totalNumOfBlocks() * sizeof(double), hipMemcpyDeviceToHost));

    for(long idx=0; idx<config.totalNumOfBlocks(); idx++) {
#ifdef __GOGO_DEBUG__
       if(config.DEBUG_LEVEL == 2)
           cout << "BLK_VARI[" << idx << "]: " << variances[idx] << endl;
#endif
       variance += variances[idx];
   }
#ifdef __GOGO_DEBUG__
   cout << endl;
#endif

    hipFree(cudaVariances);

    delete [] variances;

    return sqrt(variance);
}
